#include "host_tensor.cuh"

//DO NOT modify any code in this file
template<int N_DIMS>
void host_tensor<N_DIMS>::fill_random(){
  srand(time(NULL));
    for(size_t i=0; i<this->get_n_elems(); i++){
      this->get()[i] = float(rand()) / float(RAND_MAX) * 2.0 - 1.0;
    }
}

template<int N_DIMS>
void host_tensor<N_DIMS>::fill(float val){
    for(size_t i=0; i<this->get_n_elems(); i++){
      this->get()[i] = val;
    }
}

template<int N_DIMS>
void host_tensor<N_DIMS>::copy(const device_tensor<N_DIMS>& other){
  assert(this->get_n_elems() == other.get_n_elems());
  CHECK(hipMemcpy(this->get(), other.get(), this->get_n_elems()*sizeof(float), hipMemcpyDeviceToHost));
};

template<int N_DIMS>
void host_tensor<N_DIMS>::copy(const host_tensor<N_DIMS>& other){
  assert(this->get_n_elems() == other.get_n_elems());
  for(int i=0; i<this->get_n_elems(); i++)
    this->get()[i] = other.get()[i];
};



//Instantiate
template class host_tensor<1>;
template class host_tensor<2>;
template class host_tensor<3>;